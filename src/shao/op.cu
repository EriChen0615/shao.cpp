#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

__global__ void add_kernel(float *a, float *b, float *c) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    c[tid] = a[tid] + b[tid];
}

extern
int cuda_add(float *h_a, float *h_b, float *h_c, int n) {
    float *d_a, *d_b, *d_c;

    checkCudaErrors(hipMalloc((void **)&d_a, n * sizeof(float)) );
    checkCudaErrors(hipMalloc((void **)&d_b, n * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_c, n * sizeof(float)));

    checkCudaErrors(hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice));

    add_kernel<<<1, n>>>(d_a, d_b, d_c);
    hipGetLastError("In cuda_add: add_kernel failed");

    checkCudaErrors(hipMemcpy(h_c, d_c, n * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_a));
    checkCudaErrors(hipFree(d_b));
    checkCudaErrors(hipFree(d_c));
    return 0;
}