#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>

__global__ void add_kernel(float *a, float *b, float *c, int n) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

extern "C" int cuda_add(float *h_a, float *h_b, float *h_c, int n) {
    float *d_a, *d_b, *d_c;

    checkCudaErrors(hipMalloc((void **)&d_a, n * sizeof(float)) );
    checkCudaErrors(hipMalloc((void **)&d_b, n * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_c, n * sizeof(float)));

    checkCudaErrors(hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice));

    // Calculate grid and block dimensions
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    
    add_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    getLastCudaError("In cuda_add: add_kernel failed");

    checkCudaErrors(hipMemcpy(h_c, d_c, n * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_a));
    checkCudaErrors(hipFree(d_b));
    checkCudaErrors(hipFree(d_c));
    return 0;
}